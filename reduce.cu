#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>

#define BLOCK_SIZE 128

void random_init(int *data, int size);
void cpuTest(int& dout, int *din, int size);
__global__ void test_kernel(int *d_out, int *d_in);
__global__ void test_kernel1(int *d_out, int *d_in);
__global__ void test_kernel2(int *d_out, int *d_in);
__global__ void test_kernel3(int *d_out, int *d_in);
__global__ void test_kernel4(int *d_out, int *d_in);

int main()
{
    hipError_t hipError_t;
    hipError_t = hipSetDevice(7);
    if (hipError_t == hipSuccess)
    {
        std::cout<<"choose success"<<std::endl;   
    }
    else
    {
        std::cout<<"choose fail"<<std::endl;
    }

    const int arraySize = 16*1024*1024;
    int n_iter = 9;
    int *h_in;
    int h_out = 0;
    hipHostMalloc(&h_in,arraySize*sizeof(int));
    random_init(h_in,arraySize);

    double dur;
    clock_t start1,end1;
    start1 = clock();
    cpuTest(h_out,h_in,arraySize);
    end1 = clock();
    dur = (double)(end1 - start1);
    printf("cpu time: %f\n", (dur/CLOCKS_PER_SEC));

    int block_num = (arraySize + BLOCK_SIZE - 1) / BLOCK_SIZE / 2;
    dim3 blockDim(BLOCK_SIZE, 1, 1);
    dim3 gridDim(block_num, 1, 1);
    int block_num2 = block_num / BLOCK_SIZE / 2;
    dim3 gridDim2(block_num2, 1, 1);
    dim3 gridDim3(1, 1, 1);
    int *d_in, *d_out_L1, *d_out_L2, *d_out_L3;
    hipMalloc(&d_in, arraySize * sizeof(int));
    hipMalloc(&d_out_L1, block_num * sizeof(int));
    hipMalloc(&d_out_L2, block_num2 * sizeof(int));
    hipMalloc(&d_out_L3, 1 * sizeof(int));

    hipMemcpy(d_in, h_in, arraySize * sizeof(int), hipMemcpyDefault);

    test_kernel4 <<< gridDim, blockDim >>> (d_out_L1, d_in); // level one
    test_kernel4 <<< gridDim2, blockDim >>> (d_out_L2, d_out_L1); // level two
    test_kernel4 <<< gridDim3, blockDim >>> (d_out_L3, d_out_L2); // level three

    hipEvent_t start,end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start);
    for (size_t i = 0; i < n_iter; i++)
    {
        test_kernel4 <<< gridDim, blockDim >>> (d_out_L1, d_in); // level one
        test_kernel4 <<< gridDim2, blockDim >>> (d_out_L2, d_out_L1); // level two
        test_kernel4 <<< gridDim3, blockDim >>> (d_out_L3, d_out_L2); // level three
    }
    hipEventRecord(end);
    hipEventSynchronize(end);
    flaot msec, sec;
    hipEventElapsedTime(&msec, start, end);
    sec = msec / 1000.0 / n_iter;

    hipEventDestroy(start);
    hipEventDestroy(end);

    printf("Latency: %f\n", sec);
    
    int *h_d_out1, *h_d_out2, *h_d_out3;
    hipHostMalloc(&h_d_out1, block_num * sizeof(int));
    hipHostMalloc(&h_d_out2, block_num2 * sizeof(int));
    hipHostMalloc(&h_d_out3, sizeof(int));
    hipMemcpy(h_d_out1, d_out_L1, block_num * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_d_out2, d_out_L2, block_num2 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_d_out3, d_out_L3, sizeof(int), hipMemcpyDeviceToHost);

    int d_result1 = 0, d_result2 = 0;
    for (size_t i = 0; i < block_num; i++)
    {
        d_result1 += h_d_out1[i];
    }
    for (size_t i = 0; i < block_num2; i++)
    {
        d_result2 += h_d_out2[i];
    }

    hipHostFree(h_in);
    hipHostFree(h_d_out1);
    hipHostFree(h_d_out2);
    hipHostFree(h_d_out3);
    hipFree(d_out_L1);
    hipFree(d_out_L2);
    hipFree(d_out_L3);
    
    return 0;
}

void random_init(int *data, int size)
{
    for (size_t i = 0; i < size; i++)
    {
        data[i] = int(rand() % (10 - 1)) + 1;
    }
}

void cpuTest(int& dout, int *din, int size)
{
    for(size_t i = 0; i < size; i++)
    {
        dout += din[i];
    }
}

__global__ void test_kernel(int *d_out, int *d_in)
{
    __shared__ int sdata[BLOCK_SIZE];

    unsigned int did = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int tid = threadIdx.x;
    sdata[tid] = d_in[did];
    __syncthreads();

    for(unsigned int stride = 1; stride < blockDim.x; stride*=2)
    {
        if(tid % (2*stride) == 0)
        {
            sdata[tid] += sdata[tid + stride];
        }
        __syncthreads();
    }
    if(tid == 0)
    {
        d_out[blockIdx.x] = sdata[tid];
    }
}

__global__ void test_kernel1(int *d_out, int *d_in)
{
    __shared__ int sdata[BLOCK_SIZE];

    unsigned int did = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int tid = threadIdx.x;
    sdata[tid] = d_in[did];
    __syncthreads();

    for(unsigned int stride = 1; stride < blockDim.x; stride*=2)
    {
        int index = stride * 2 * tid;
        if(index < blockDim.x)
        {
            sdata[index] += sdata[index + stride];
        }
        __syncthreads();
    }
    if(tid == 0)
    {
        d_out[blockIdx.x] = sdata[tid];
    }
}

__global__ void test_kernel2(int *d_out, int *d_in)
{
    __shared__ int sdata[BLOCK_SIZE];

    unsigned int did = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int tid = threadIdx.x;
    sdata[tid] = d_in[did];
    __syncthreads();

    for(unsigned int stride = blockDim.x/2; stride > 0; stride>>=1)
    {
        if(tid < stride)
        {
            sdata[tid] += sdata[tid + stride];
        }
        __syncthreads();
    }
    if(tid == 0)
    {
        d_out[blockIdx.x] = sdata[tid];
    }
}

__global__ void test_kernel3(int *d_out, int *d_in)
{
    __shared__ int sdata[BLOCK_SIZE];

    unsigned int did = (blockDim.x*2) * blockIdx.x + threadIdx.x;
    unsigned int tid = threadIdx.x;
    sdata[tid] = d_in[did] + d_in[did + blockDim.x];
    __syncthreads();

   for(unsigned int stride = blockDim.x/2; stride > 0; stride>>=1)
    {
        if(tid < stride)
        {
            sdata[tid] += sdata[tid + stride];
        }
        __syncthreads();
    }
    if(tid == 0)
    {
        d_out[blockIdx.x] = sdata[tid];
    }
}

__global__ void test_kernel4(int *d_out, int *d_in)
{
    __shared__ int sdata[BLOCK_SIZE];

    unsigned int did = (blockDim.x*2) * blockIdx.x + threadIdx.x;
    unsigned int tid = threadIdx.x;
    sdata[tid] = d_in[did] + d_in[did + blockDim.x];
    __syncthreads();

   for(unsigned int stride = blockDim.x/2; stride > 32; stride>>=1)
    {
        if(tid < stride)
        {
            sdata[tid] += sdata[tid + stride];
        }
        __syncthreads();
    }
    if(tid < 32)
    {
        sdata[tid] += sdata[tid + 32];
        sdata[tid] += sdata[tid + 16];
        sdata[tid] += sdata[tid + 8];
        sdata[tid] += sdata[tid + 4];
        sdata[tid] += sdata[tid + 2];
        sdata[tid] += sdata[tid + 1];
    }
    if(tid == 0)
    {
        d_out[blockIdx.x] = sdata[tid];
    }
}