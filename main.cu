#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include "hip/hip_runtime_api.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <iostream>
#include <time.h>

void random_init(float *data, size_t size);
void cpuSgemm(float *a, float *b, float *c, const int M, const int N, const int K);
__global__ void naiveSgemm(float *a, float *b, float *c, const int M, const int N, const int K);
__global__ void sgemm_kernel(float *A, float *B, float *C, const int M, const int N, const int K);

int main()
{
	int m = 512;
	int n = 512;
	int k = 512;
	int n_iter = 9;

	hipError_t hipError_t;
	hipError_t = hipSetDevice(7);
	if(hipError_t == hipSuccess)
	{
		std::cout << "choose success" << std::endl;
	}
	else
	{
		std::cout << "choose fail" << std::endl;
	}
	int deviceId; 
	hipError_t = hipGetDevice(&deviceId);

	float *h_A, *h_B, *h_C, *h_d_C;
	hipHostMalloc(&h_A, m*k*sizeof(float));
	hipHostMalloc(&h_B, k*n*sizeof(float));
	hipHostMalloc(&h_C, m*n*sizeof(float));
	hipHostMalloc(&h_d_C, m*n*sizeof(float));
	random_init(h_A, m*k);
	random_init(h_B, k*n);

	double dur;
	clock_t start1,endl;
	start1 = clock();
	cpuSgemm(h_A, h_B, h_C, m, n, k);
	end1 = clock();
	dur = (double)(end1 - start1);
	printf("cpu time: %f\n",(dur/CLOCLS_PER_SEC));

	float *d_A, *d_B, *d_C;
	hipMalloc(&d_A, m*k*sizeof(float));
	hipMalloc(&d_B, k*n*sizeof(float));
	hipMalloc(&d_C, m*n*sizeof(float));

	hipMemcpy(d_A, h_A, m*k*sizeof(float),hipMemcpyDefault);
	hipMemcpy(d_B, h_B, k*n*sizeof(float),hipMemcpyDefault);
	
	dim3 blockDim(32,32);
	dim3 gridDime((n+32-1)/32,(m+32-1)/32);

	naiveSgemm<<<gridDim,blockDim>>>(d_A,d_B,d_C,m,n,k);
	sgemm_kernel<<<gridDim,blockDim>>>(d_A,d_B,d_C,m,n,k);

	hipEvent_t start,end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start);

	for(size_t i = 0; i < n_iter; i++)
	{
		naiveSgemm<<<gridDim,blockDim>>>(d_A,d_B,d_C,m,n,k);
		sgemm_kernel<<<gridDim,blockDim>>>(d_A,d_B,d_C,m,n,k);
	}

	hipEventRecord(end);
	hipEventSynchronize(end);
	float msec, sec;
	hipEventElapsedTime(&msec, start, end);
	sec = msec / 1000.0 / n_iter;

	hipEventDestroy(start);
	hipEventDestroy(end);

	printf("Latency: %f\n",sec);

	hipMemcpy(h_d_C, d_C, m*n*sizeof(float),hipMemcpyDeviceToHost);
	float abserror = 0.0;
	for(size_t i = 0; i < m*n; i++)
	{
		float temperror = abs(h_d_C[i]-h_C[i]);
		if(temperror > abserror)
		{
			abserror  = temperror;
		}
	}

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipHostFree(h_A);
	hipHostFree(h_B);
	hipHostFree(h_C);
	hipHostFree(h_d_C);

	return 0;
}

void randomA_init(float *data, size_t size)
{
	for(size_t i = 0; i < size; i++)
	{
		data[i] = float(rand()) / RAND_MAX;
	}
}

void cpuSgemm(float *a, float *b, float *c, const int M, const int N, const int K)
{
	for(size_t m = 0; m < M; m++)
	{
		for(size_t n = 0; n < N; n++)
		{
			float psum = 0.0;
			for(size_t k = 0; k < K; k++)
			{
				psum += a[k + m*K] * b[n + k*N];
			}
			c[n+m*N] = psum;
		}
	}
}

__global__ void naiveSgemm(float *a, float *b, float *c, const int M, const int N, const int K)
{
	int n = blockIdx.x * blockDim.x + threadIdx.x;
	int m = blockIdx.y * blockDim.y + threadIdx.y;
	if(m < M && n < N)
	{
		float psum = 0.0;
		for(size_t k = 0; k < K; k++)
		{
			psum += a[k + m*K] * b[n+k*N];
		}
		c[n + m*N] = psum;
	}
}

__global__ void sgemm_kernel(float *A, float *B, float *C, const int M, const int N, const int K)
{
	// blocked matrix multiply
	__shared__ float tileA[32][32];
	__shared__ float tileB[32][32];

	int tx = threadIdx.x,ty = threadIdx.y;
	int n = blockIdx.x * blockDim.x + threadIdx.x;
	int m = blockIdx.y * blockDim.y + threadIdx.y;
	if(n >=N || m>=M)
	{
		return;
	}

	float psum = 0.0;
	for(int idx_tile = 0; idx_tile < K/32; idx_tile++)
	{
		tileA[ty][tx] = A[m*K + tx + idx_tile*32];
		tileB[ty][tx] = B[(idx_tile * 32 + ty) * K + n];
		__syncthreads();
		for(int k = 0; k < 32; k++)
		{
			psum += tileA[ty][k] * tileB[k][tx];
		}
		__syncthreads();
	}
	C[n + m*N] = psum;
}